#include "hip/hip_runtime.h"
#include <stdio.h>
#include "sha1.cu"

#define MESSAGE_SIZE 1
#define HASH_SIZE 20
#define MAX_ARITY 3

//use the current levels arity
#define getChildIdx(index,start,end,arity)\
	((index-start)*arity)+end+1;

//the arity is the parents arity 
#define getParentIdx(index,start,end,arity)\
	start - ((end-start+1)/arity - ((index - start)/arity))

typedef unsigned char UCHAR;
struct node {
	UCHAR hash[HASH_SIZE];
	uint8_t hashed;
};
typedef struct node m_node;


void printHash(const UCHAR *hash){
	for (int i=0;i<HASH_SIZE;i++)
		printf("%02x",hash[i]);
	printf("\n");
}
void printTree(
	const m_node   *nodes,
	const uint64_t *startIdx, 
	const uint64_t *endIdx,
	uint8_t         height 
)
{

	printf("level: %d,",  height);
	printf("nodes: %ld\n", endIdx[height] - startIdx[height] + 1);
	for (uint64_t i=startIdx[height];i<=endIdx[height];i++)
		printHash(nodes[i].hash);
	printf("\n");
	if (height == 0)
		return;
	printTree(nodes,startIdx,endIdx,height-1);
}

__global__ 
void hashTreeP 
(
	m_node   *nodes,
	uint64_t *startIdx,
	uint64_t *endIdx,
	uint8_t  *arities,
	uint8_t  height,
	const UCHAR    *message
)
{
	//create a buffer that will be used several times;
	UCHAR buffer[HASH_SIZE*MAX_ARITY];
	//find location and set currIdx and childIdx
	uint64_t curr = blockIdx.x * blockDim.x + threadIdx.x;
	curr += startIdx[1];
	if (curr>endIdx[1])
		return;
	uint64_t childIdx = getChildIdx(curr,startIdx[1],endIdx[1],arities[1]);
	//hash children and store the concatenated results in the buff
	for (uint8_t i = 0;i<arities[1];i++){
		SHA1((buffer+(i*HASH_SIZE)),message,MESSAGE_SIZE);
		memcpy(nodes[childIdx+i].hash,(buffer+(i*HASH_SIZE)),HASH_SIZE);
		nodes[childIdx+i].hashed = 1;
	}
	//hash the concatenations together
	SHA1(nodes[curr].hash,buffer,HASH_SIZE*arities[1]);
	nodes[curr].hashed = 1;
	//only one sibling moves to the parent
	if (curr%arities[2] != 0)
		return;

	//go to the parent node, now at level 2, save the child index
	curr = getParentIdx(curr,startIdx[1],endIdx[1],arities[2]);

	//iterate through the tree
	for (uint8_t i=2;i<=height;i++){
		childIdx = getChildIdx(curr,startIdx[i],endIdx[i],arities[i]);
		//wait for the children to be hashed
		uint8_t flag;
		while(1){
			flag = 0;
			for (uint8_t j=0;j<arities[i];j++){
				if (nodes[childIdx+j].hashed==1)
					flag++;
			}
			if(flag == arities[i])
				break;
		}
		for (int j=0;j<arities[i];j++){
			if (nodes[childIdx+j].hashed == 0)
				printf("oh no\n");
		}
		//concat the children
		for (uint8_t j=0;j<arities[i];j++)
			memcpy((buffer+(j*HASH_SIZE)),nodes[childIdx+j].hash,HASH_SIZE);
		//hash the concatenations
		SHA1(nodes[curr].hash,buffer,HASH_SIZE*arities[i]);
		//only one sibling continues
		if (curr == 0  | curr%arities[i+1] != 0)
			return;
		//move onto next level
		curr = getParentIdx(curr,startIdx[i],endIdx[i],arities[i+1]);
	}
}

int main(int argc,char **argv){
	if (argc != 2){
		printf("enter correct args\n");
		return 0;
	}

	//calculate the arities
	const uint64_t num_blocks = atoi(argv[1]);
	const uint8_t  height     = ceil(log10(num_blocks)/log10(3));
	const uint8_t  num_twos   = log10(num_blocks/pow(3,height))/log10(2.0f/3.0f);
	const uint8_t  num_threes = height - num_twos;
	const uint64_t num_leaves = pow(2,num_twos) * pow(3,num_threes);
	uint8_t  arities[height+1];
	arities[0] = 0;
	for (uint8_t i = 1;i<=num_threes;i++)
		arities[i] = 3;
	for (uint8_t i = num_threes+1;i<=height;i++)
		arities[i] = 2;

	//determine start and end index for each level
	//they are is used to navigate the tree
	uint64_t startIdx[height+1];
	uint64_t endIdx[height+1];
	uint64_t nodes_at_level;
	for (uint64_t i = height;i>=0; i--){
		if (i == height){
			startIdx[i] = 0;
			nodes_at_level = 1;
		}
		else{
			startIdx[i] = endIdx[i+1] + 1;
			nodes_at_level = (endIdx[i+1] - startIdx[i+1] + 1 ) * arities[i+1];
		}
		endIdx[i] = startIdx[i] + nodes_at_level - 1;
		if (i == 0)
			break;
	}


	//create the message string
	UCHAR message[MESSAGE_SIZE];
	for (int i=0;i<MESSAGE_SIZE;i++)
		message[i] = 'a';
	//create the nodes tree
	m_node *nodes = (m_node*)malloc((endIdx[0]+1)*sizeof(m_node));
	for (uint64_t i =0;i<=endIdx[0];i++)
		nodes[i].hashed = 0;

	//allocate CudaMemory
	m_node   *d_nodes;
	UCHAR    *d_message;
	uint64_t *d_startIdx,*d_endIdx;
	uint8_t  *d_arities;

	hipMalloc(&d_message,MESSAGE_SIZE*sizeof(UCHAR));
	hipMemcpy(d_message, message,MESSAGE_SIZE*sizeof(UCHAR),
		hipMemcpyHostToDevice);

	hipMalloc(&d_nodes,(endIdx[0]+1)*sizeof(m_node));

	hipMalloc(&d_startIdx,(height+1)*sizeof(uint64_t));
	hipMemcpy(d_startIdx, startIdx,(height+1)*sizeof(uint64_t),
		hipMemcpyHostToDevice);

	hipMalloc(&d_endIdx,(height+1)*sizeof(uint64_t));
	hipMemcpy(d_endIdx, endIdx,(height+1)*sizeof(uint64_t),
		hipMemcpyHostToDevice);

	hipMalloc(&d_arities,(height+1)*sizeof(uint8_t));
	hipMemcpy(d_arities,arities,(height+1)*sizeof(uint8_t),
		hipMemcpyHostToDevice);

	//execute kernel function and extract the memory
	printf("Kernel Working... \n");
	uint64_t N = endIdx[1] - startIdx[1] + 1;
	hashTreeP<<< ( (N+255)/256 ) , 256 >>>(
	 	d_nodes,
	 	d_startIdx,
	 	d_endIdx,
	 	d_arities,
	 	height,
	 	d_message
	);
	hipDeviceSynchronize();
	hipMemcpy(nodes,d_nodes,(endIdx[0]+1)*sizeof(m_node),
		hipMemcpyDeviceToHost);
	
	printTree(nodes,startIdx,endIdx,height);

	hipFree(d_nodes);
	hipFree(d_message);
	hipFree(d_endIdx);
	hipFree(d_startIdx);
	hipFree(d_arities);
	free(nodes);
	return 0;
}
