#include "hip/hip_runtime.h"
#include <stdint.h>
#include <stdlib.h>
#include <stdio.h>
#include <math.h>
#include <string.h>

#define HASH_SIZE 20

typedef struct
{
	unsigned char hash[HASH_SIZE];
} m_node;
typedef struct 
{
	m_node *nodes;
	uint8_t *arities;
	uint64_t *offsets;
	uint64_t *startIdx;
	uint64_t *endIdx;
	uint8_t height;
	unsigned char *message;
	uint64_t messageSize;
} m_tree;


//helper function for create*Tree functions
void configureTree(m_tree *tree)
{
	//fill offsets array
	tree->offsets[1] = 1;
	for (uint8_t i=2;i<=tree->height;i++)
		tree->offsets[i] = tree->arities[i]*tree->offsets[i-1];

	//fill startIdx and endIdx
	uint64_t nodes_at_level;
	for (uint64_t i = tree->height;i>=0; i--){
		if (i == tree->height){
			tree->startIdx[i] = 0;
			nodes_at_level = 1;
		}
		else{
			tree->startIdx[i] = tree->endIdx[i+1] + 1;
			nodes_at_level = (tree->endIdx[i+1] - tree->startIdx[i+1] + 1) *
			tree->arities[i+1];
		}
		tree->endIdx[i] = tree->startIdx[i] + nodes_at_level - 1;
		if (i == 0)
			break;
	}
}

//allocates memory for tree struct data
//computes tree struct's parameter data
void createOptimizedTree
(
	m_tree* tree,
	uint64_t numBlocks,
	uint64_t messageSize
)
{
	tree->messageSize  = messageSize;
	tree->height       = ceil(log10(numBlocks)/log10(3));
	uint8_t  numTwos   = log10(numBlocks/pow(3,tree->height))/log10(2.0f/3.0f);
	uint8_t  numThrees = tree->height - numTwos;
	tree->arities      = (uint8_t  *)malloc((tree->height+1)*sizeof(uint8_t));
	tree->offsets      = (uint64_t *)malloc((tree->height+1)*sizeof(uint64_t));
	tree->startIdx     = (uint64_t *)malloc((tree->height+1)*sizeof(uint64_t));
	tree->endIdx       = (uint64_t *)malloc((tree->height+1)*sizeof(uint64_t));
	tree->message      = (unsigned char *)malloc(messageSize*sizeof(unsigned char));
	
	//fill arities array
	tree->arities[0] = 0;
	for (uint8_t i = 1;i<=numThrees;i++)
		tree->arities[i] = 3;
	for (uint8_t i = numThrees+1;i<=tree->height;i++)
		tree->arities[i] = 2;

	//find offsets, startIDx, endIdx
	configureTree(tree);
	tree->nodes = (m_node *)malloc(sizeof(m_node)*(tree->endIdx[0]+1));
	
	//fill message string - arbitray message
	for (uint64_t i=0;i<messageSize;i++)
		tree->message[i] = 'a';
}
void createBinaryTree
(
	m_tree* tree,
	uint64_t numBlocks,
	uint64_t messageSize
)
{
	tree->messageSize = messageSize;
	tree->height      = ceil(log10(numBlocks)/log10(2));
	tree->arities     = (uint8_t  *)malloc((tree->height+1)*sizeof(uint8_t));
	tree->offsets     = (uint64_t *)malloc((tree->height+1)*sizeof(uint64_t));
	tree->startIdx    = (uint64_t *)malloc((tree->height+1)*sizeof(uint64_t));
	tree->endIdx      = (uint64_t *)malloc((tree->height+1)*sizeof(uint64_t));
	tree->message     = (unsigned char *)malloc(messageSize*sizeof(unsigned char));
	
	//fill arities array
	tree->arities[0] = 0;
	for (uint8_t i = 1;i<=tree->height;i++)
		tree->arities[i] = 2;

	//find offsets, startIDx, endIdx
	configureTree(tree);
	tree->nodes = (m_node *)malloc(sizeof(m_node)*(tree->endIdx[0]+1));

	//fill message string - arbitray message
	for (uint64_t i=0;i<messageSize;i++)
		tree->message[i] = 'a';
}


//memory management functions
void cudaCopyTree(m_tree *d_tree,m_tree *h_tree)
{
	d_tree->height       = h_tree->height;
	d_tree->messageSize  = h_tree->messageSize;
	hipMalloc(&d_tree->message,d_tree->messageSize*sizeof(char));
	hipMemcpy(
		d_tree->message, 
		h_tree->message,
		d_tree->messageSize*sizeof(char),
		hipMemcpyHostToDevice);

	hipMalloc(&d_tree->nodes,(h_tree->endIdx[1]-h_tree->startIdx[1]+1)*sizeof(m_node));

	hipMalloc(&d_tree->arities,(h_tree->height+1)*sizeof(char));
	hipMemcpy(
		d_tree->arities,
		h_tree->arities,
		(h_tree->height+1)*sizeof(uint8_t),
		hipMemcpyHostToDevice
	);

	hipMalloc(&d_tree->offsets,(h_tree->height+1)*sizeof(uint64_t));
	hipMemcpy(
			d_tree->offsets,
			h_tree->offsets,
			(h_tree->height+1)*sizeof(uint64_t),
			hipMemcpyHostToDevice
	);
}
void freeTree(m_tree *tree)
{
	free(tree->nodes);
	free(tree->arities);
	free(tree->offsets);
	free(tree->startIdx);
	free(tree->endIdx);
	free(tree->message);
}
void cudaFreeTree(m_tree *tree)
{
	hipFree(tree->nodes);
	hipFree(tree->message);
	hipFree(tree->offsets);
	hipFree(tree->arities);
}

//parallel, GPU implementation of the merkle tree
//capable of generating merkle roots of the variable tree modes 
__global__ 
void hashTreeP 
(
	m_node   *nodes,
	uint64_t N,
	uint8_t  *arities,
	uint64_t *offsets,
	uint8_t  height,
	const unsigned char    *message
)
{
	unsigned char buffer[HASH_SIZE*MAX_ARITY];
	uint16_t thread = threadIdx.x;
	uint16_t block_size = blockDim.x;

	//calculate the message's hash
	for (uint64_t idx=thread; idx<N; idx+=block_size){
		for (uint8_t i=0;i<arities[1];i++)
			SHA1((buffer+(i*HASH_SIZE)),message,MESSAGE_SIZE);
		SHA1(nodes[idx].hash,buffer,HASH_SIZE*arities[1]);
	}
	__syncthreads();

	//parallel reduction begins, only one child will proceed
	//through each height level
	for (uint8_t i=2;i<=height;i++){
		for (uint64_t idx=thread;idx<N;idx+=block_size){
			if (idx%offsets[i]==0){
				for (uint8_t j=0;j<arities[i];j++){
					memcpy((buffer+(j*HASH_SIZE)),
							nodes[idx+j*offsets[i-1]].hash,
							HASH_SIZE);
				}
				SHA1(nodes[idx].hash,buffer,HASH_SIZE*arities[i]);
			}
		}
		__syncthreads();
	}
}


//a serial merkle root generator for error checking
//the macros are used only in hashTreeS
//use the current levels arity
#define getChildIdx(index,start,end,arity)\
	((index-start)*arity)+end+1;
//the arity is the parents arity 
#define getParentIdx(index,start,end,arity)\
	start - ((end-start+1)/arity - ((index - start)/arity))
void hashTreeS (m_tree *tree)
{
	unsigned char *tmp = (unsigned char*)malloc(HASH_SIZE*3*sizeof(unsigned char));
	for (int i = tree->startIdx[0];i<=tree->endIdx[0];i++){
		SHA1(tree->nodes[i].hash,tree->message,tree->messageSize);
	}
	uint64_t childIdx;
	for (uint64_t i=1;i<=tree->height;i++){
		for (uint64_t j=tree->startIdx[i];j<=tree->endIdx[i];j++){
			childIdx = getChildIdx(j,tree->startIdx[i],tree->endIdx[i],tree->arities[i]);
			for (uint64_t k = childIdx;k<childIdx+tree->arities[i];k++)
				memcpy((tmp+(k-childIdx)*HASH_SIZE),tree->nodes[k].hash,HASH_SIZE);
			SHA1(tree->nodes[j].hash,tmp,HASH_SIZE*tree->arities[i]);
		}
	}
	free(tmp);
}

